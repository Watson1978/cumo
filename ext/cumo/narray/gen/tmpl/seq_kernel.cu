#include "hip/hip_runtime.h"
<% if is_int && !is_object %>
typedef double seq_data_t;
<% else %>
typedef dtype seq_data_t;
<% end %>

<% if is_object %>
typedef size_t seq_count_t;
<% else %>
typedef double seq_count_t;
<% end %>

<% unless is_object %>
__global__ void <%="cumo_#{c_iter}_index_kernel"%>(char *p1, size_t* idx1, seq_data_t beg, seq_data_t step, seq_count_t c, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        dtype x = f_seq(beg,step,c+i);
        *(dtype*)(p1+idx1[i]) = x;
    }
}

__global__ void <%="cumo_#{c_iter}_stride_kernel"%>(char *p1, size_t s1, seq_data_t beg, seq_data_t step, seq_count_t c, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        dtype x = f_seq(beg,step,c+i);
        *(dtype*)(p1+(i*s1)) = x;
    }
}

void <%="cumo_#{c_iter}_index_kernel_launch"%>(char *p1, size_t* idx1, seq_data_t beg, seq_data_t step, seq_count_t c, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    <%="cumo_#{c_iter}_index_kernel"%><<<grid_dim, block_dim>>>(p1,idx1,beg,step,c,n);
}

void <%="cumo_#{c_iter}_stride_kernel_launch"%>(char *p1, ssize_t s1, seq_data_t beg, seq_data_t step, seq_count_t c, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    <%="cumo_#{c_iter}_stride_kernel"%><<<grid_dim, block_dim>>>(p1,s1,beg,step,c,n);
}
<% end %>
