#include "hip/hip_runtime.h"
<% unless type_name == 'robject' %>
__global__ void <%="cumo_#{c_iter}_kernel"%>(char *p1, char *p2, char *p3, ssize_t s1, ssize_t s2, ssize_t s3, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        *(dtype*)(p3 + (i * s3)) = m_pow(*(dtype*)(p1 + (i * s1)), *(dtype*)(p2 + (i * s2)));
    }
}

__global__ void <%="cumo_#{c_iter}_int32_kernel"%>(char *p1, char *p2, char *p3, ssize_t s1, ssize_t s2, ssize_t s3, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        *(dtype*)(p3 + (i * s3)) = m_pow_int(*(dtype*)(p1 + (i * s1)), *(int32_t*)(p2 + (i * s2)));
    }
}

void <%="cumo_#{c_iter}_kernel_launch"%>(char *p1, char *p2, char *p3, ssize_t s1, ssize_t s2, ssize_t s3, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    <%="cumo_#{c_iter}_kernel"%><<<grid_dim, block_dim>>>(p1,p2,p3,s1,s2,s3,n);
}

void <%="cumo_#{c_iter}_int32_kernel_launch"%>(char *p1, char *p2, char *p3, ssize_t s1, ssize_t s2, ssize_t s3, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    <%="cumo_#{c_iter}_int32_kernel"%><<<grid_dim, block_dim>>>(p1,p2,p3,s1,s2,s3,n);
}
<% end %>
