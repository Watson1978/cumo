#include "hip/hip_runtime.h"
<% unless type_name == 'robject' %>
__global__ void <%="#{c_iter}_stride_kernel"%>(char *p1, char *p2, char *p3, char *p4, ssize_t s1, ssize_t s2, ssize_t s3, ssize_t s4, size_t n)
{
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        m_<%=name%>(*(dtype*)(p1+(i*s1)),*(dtype*)(p2+(i*s2)),*(dtype*)(p3+(i*s3)), *(dtype*)(p4+(i*s4)));
    }
}

void <%="#{c_iter}_stride_kernel_launch"%>(char *p1, char *p2, char *p3, char *p4, ssize_t s1, ssize_t s2, ssize_t s3, ssize_t s4, size_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="#{c_iter}_stride_kernel"%><<<gridDim, blockDim>>>(p1,p2,p3,p4,s1,s2,s3,s4,n);
}
<% end %>
