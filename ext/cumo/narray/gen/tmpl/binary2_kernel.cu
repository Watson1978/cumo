#include "hip/hip_runtime.h"
<% unless type_name == 'robject' %>
__global__ void <%="cumo_#{c_iter}_stride_kernel"%>(char *p1, char *p2, char *p3, char *p4, ssize_t s1, ssize_t s2, ssize_t s3, ssize_t s4, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        m_<%=name%>(*(dtype*)(p1+(i*s1)),*(dtype*)(p2+(i*s2)),*(dtype*)(p3+(i*s3)), *(dtype*)(p4+(i*s4)));
    }
}

void <%="cumo_#{c_iter}_stride_kernel_launch"%>(char *p1, char *p2, char *p3, char *p4, ssize_t s1, ssize_t s2, ssize_t s3, ssize_t s4, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    <%="cumo_#{c_iter}_stride_kernel"%><<<grid_dim, block_dim>>>(p1,p2,p3,p4,s1,s2,s3,s4,n);
}
<% end %>
