#include "hip/hip_runtime.h"
<% unless c_iter.include? 'robject' %>
__global__ void <%="cumo_#{c_iter}_index_index_kernel"%>(char *p1, char *p2, size_t *idx1, size_t *idx2, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        *(dtype*)(p1 + idx1[i]) = <%=macro%>(*(<%=dtype%>*)(p2 + idx2[i]));
    }
}

__global__ void <%="cumo_#{c_iter}_stride_index_kernel"%>(char *p1, char *p2, ssize_t s1, size_t *idx2, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        *(dtype*)(p1 + (i * s1)) = <%=macro%>(*(<%=dtype%>*)(p2 + idx2[i]));
    }
}

__global__ void <%="cumo_#{c_iter}_index_stride_kernel"%>(char *p1, char *p2, size_t *idx1, ssize_t s2, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        *(dtype*)(p1 + idx1[i]) = <%=macro%>(*(<%=dtype%>*)(p2 + (i * s2)));
    }
}

//<% ((0..opt_indexer_ndim).to_a << '').each do |idim| %>
__global__ void <%="cumo_#{c_iter}_stride_stride_kernel_dim#{idim}"%>(na_iarray_t a1, na_iarray_t a2, na_indexer_t indexer)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < indexer.total_size; i += blockDim.x * gridDim.x) {
        cumo_na_indexer_set_dim<%=idim%>(&indexer, i);
        char* p1 = cumo_na_iarray_at_dim<%=idim%>(&a1, &indexer);
        char* p2 = cumo_na_iarray_at_dim<%=idim%>(&a2, &indexer);
        *(dtype*)(p1) = <%=macro%>(*(<%=dtype%>*)(p2));
    }
}
//<% end %>

void <%="cumo_#{c_iter}_index_index_kernel_launch"%>(char *p1, char *p2, size_t *idx1, size_t *idx2, uint64_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="cumo_#{c_iter}_index_index_kernel"%><<<gridDim, blockDim>>>(p1,p2,idx1,idx2,n);
}

void <%="cumo_#{c_iter}_stride_index_kernel_launch"%>(char *p1, char *p2, ssize_t s1, size_t *idx2, uint64_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="cumo_#{c_iter}_stride_index_kernel"%><<<gridDim, blockDim>>>(p1,p2,s1,idx2,n);
}

void <%="cumo_#{c_iter}_index_stride_kernel_launch"%>(char *p1, char *p2, size_t *idx1, ssize_t s2, uint64_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="cumo_#{c_iter}_index_stride_kernel"%><<<gridDim, blockDim>>>(p1,p2,idx1,s2,n);
}

void <%="cumo_#{c_iter}_stride_stride_kernel_launch"%>(na_iarray_t* a1, na_iarray_t* a2, na_indexer_t* indexer)
{
    size_t gridDim = get_gridDim(indexer->total_size);
    size_t blockDim = get_blockDim(indexer->total_size);
    switch (indexer->ndim) {
    <% (0..opt_indexer_ndim).each do |idim| %>
    case <%=idim%>:
        <%="cumo_#{c_iter}_stride_stride_kernel_dim#{idim}"%><<<gridDim, blockDim>>>(*a1,*a2,*indexer);
        break;
    <% end %>
    default:
        <%="cumo_#{c_iter}_stride_stride_kernel_dim"%><<<gridDim, blockDim>>>(*a1,*a2,*indexer);
        break;
    }
}

<% end %>
