#include "hip/hip_runtime.h"
<% unless type_name == 'robject' %>
__global__ void <%="cumo_#{c_iter}_stride_kernel"%>(char*ptr, ssize_t s0, ssize_t s1, ssize_t kofs, dtype data, uint64_t n0, uint64_t n1, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        uint64_t i0 = i / n1;
        uint64_t i1 = i - (i0 * n1);
        *(dtype*)(ptr + (i0*s0) + (i1*s1)) = (i0+kofs==i1) ? data : m_zero;
    }
}

void <%="cumo_#{c_iter}_stride_kernel_launch"%>(char *ptr, ssize_t s0, ssize_t s1, ssize_t kofs, dtype data, uint64_t n0, uint64_t n1)
{
    uint64_t n = n0 * n1;
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    <%="cumo_#{c_iter}_stride_kernel"%><<<grid_dim, block_dim>>>(ptr,s0,s1,kofs,data,n0,n1,n);
}
<% end %>

