#include "hip/hip_runtime.h"
<% if type_name == 'robject' || name == 'map' %>
<% else %>
__global__ void <%="cumo_#{c_iter}_index_index_kernel"%>(char *p1, char *p2, size_t *idx1, size_t *idx2, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        *(dtype*)(p2 + idx2[i]) = m_<%=name%>(*(dtype*)(p1 + idx1[i]));
    }
}

__global__ void <%="cumo_#{c_iter}_index_stride_kernel"%>(char *p1, char *p2, size_t *idx1, ssize_t s2, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        *(dtype*)(p2 + (i * s2)) = m_<%=name%>(*(dtype*)(p1 + idx1[i]));
    }
}

__global__ void <%="cumo_#{c_iter}_stride_index_kernel"%>(char *p1, char *p2, ssize_t s1, size_t *idx2, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        *(dtype*)(p2 + idx2[i]) = m_<%=name%>(*(dtype*)(p1 + (i * s1)));
    }
}

__global__ void <%="cumo_#{c_iter}_stride_stride_kernel"%>(char *p1, char *p2, ssize_t s1, ssize_t s2, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        *(dtype*)(p2 + (i * s2)) = m_<%=name%>(*(dtype*)(p1 + (i * s1)));
    }
}

__global__ void <%="cumo_#{c_iter}_contiguous_kernel"%>(char *p1, char *p2, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        ((dtype*)p2)[i] = m_<%=name%>(((dtype*)p1)[i]);
    }
}

void <%="cumo_#{c_iter}_index_index_kernel_launch"%>(char *p1, char *p2, size_t *idx1, size_t *idx2, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    <%="cumo_#{c_iter}_index_index_kernel"%><<<grid_dim, block_dim>>>(p1,p2,idx1,idx2,n);
}

void <%="cumo_#{c_iter}_index_stride_kernel_launch"%>(char *p1, char *p2, size_t *idx1, ssize_t s2, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    <%="cumo_#{c_iter}_index_stride_kernel"%><<<grid_dim, block_dim>>>(p1,p2,idx1,s2,n);
}

void <%="cumo_#{c_iter}_stride_index_kernel_launch"%>(char *p1, char *p2, ssize_t s1, size_t *idx2, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    <%="cumo_#{c_iter}_stride_index_kernel"%><<<grid_dim, block_dim>>>(p1,p2,s1,idx2,n);
}

void <%="cumo_#{c_iter}_stride_stride_kernel_launch"%>(char *p1, char *p2, ssize_t s1, ssize_t s2, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    <%="cumo_#{c_iter}_stride_stride_kernel"%><<<grid_dim, block_dim>>>(p1,p2,s1,s2,n);
}

void <%="cumo_#{c_iter}_contiguous_kernel_launch"%>(char *p1, char *p2, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    <%="cumo_#{c_iter}_contiguous_kernel"%><<<grid_dim, block_dim>>>(p1,p2,n);
}
<% end %>
