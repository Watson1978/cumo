#include "hip/hip_runtime.h"
<% unless c_iter.include? 'robject' %>
__global__ void <%="cumo_#{c_iter}_index_kernel"%>(char *p1, size_t *idx1, dtype* z, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        *(dtype*)(p1 + idx1[i]) = z[i];
    }
}

__global__ void <%="cumo_#{c_iter}_stride_kernel"%>(char *p1, ssize_t s1, dtype* z, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        *(dtype*)(p1 + (i * s1)) = z[i];
    }
}

__global__ void <%="cumo_#{c_iter}_index_scalar_kernel"%>(char *p1, size_t *idx1, dtype z, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        *(dtype*)(p1 + idx1[i]) = z;
    }
}

__global__ void <%="cumo_#{c_iter}_stride_scalar_kernel"%>(char *p1, ssize_t s1, dtype z, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        *(dtype*)(p1 + (i * s1)) = z;
    }
}

void <%="cumo_#{c_iter}_index_kernel_launch"%>(char *p1, size_t *idx1, dtype* z, uint64_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="cumo_#{c_iter}_index_kernel"%><<<gridDim, blockDim>>>(p1,idx1,z,n);
}

void <%="cumo_#{c_iter}_stride_kernel_launch"%>(char *p1, ssize_t s1, dtype* z, uint64_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="cumo_#{c_iter}_stride_kernel"%><<<gridDim, blockDim>>>(p1,s1,z,n);
}

void <%="cumo_#{c_iter}_index_scalar_kernel_launch"%>(char *p1, size_t *idx1, dtype z, uint64_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="cumo_#{c_iter}_index_scalar_kernel"%><<<gridDim, blockDim>>>(p1,idx1,z,n);
}

void <%="cumo_#{c_iter}_stride_scalar_kernel_launch"%>(char *p1, ssize_t s1, dtype z, uint64_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="cumo_#{c_iter}_stride_scalar_kernel"%><<<gridDim, blockDim>>>(p1,s1,z,n);
}

<% end %>
