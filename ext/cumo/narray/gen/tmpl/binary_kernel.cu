#include "hip/hip_runtime.h"
<% unless type_name == 'robject' %>

<% ((0..opt_indexer_ndim).to_a << '').each do |idim| %>
__global__ void <%="cumo_#{c_iter}_kernel_dim#{idim}"%>(na_iarray_t a1, na_iarray_t a2, na_iarray_t a3, na_indexer_t indexer)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < indexer.total_size; i += blockDim.x * gridDim.x) {
        cumo_na_indexer_set_dim<%=idim%>(&indexer, i);
        char* p1 = cumo_na_iarray_at_dim<%=idim%>(&a1, &indexer);
        char* p2 = cumo_na_iarray_at_dim<%=idim%>(&a2, &indexer);
        char* p3 = cumo_na_iarray_at_dim<%=idim%>(&a3, &indexer);
        *(dtype*)(p3) = m_<%=name%>(*(dtype*)(p1),*(dtype*)(p2));
    }
}
<% end %>

void <%="cumo_#{c_iter}_kernel_launch"%>(na_iarray_t* a1, na_iarray_t* a2, na_iarray_t* a3, na_indexer_t* indexer)
{
    size_t gridDim = get_gridDim(indexer->total_size);
    size_t blockDim = get_blockDim(indexer->total_size);
    switch (indexer->ndim) {
    <% (0..opt_indexer_ndim).each do |idim| %>
    case <%=idim%>:
        <%="cumo_#{c_iter}_kernel_dim#{idim}"%><<<gridDim, blockDim>>>(*a1,*a2,*a3,*indexer);
        break;
    <% end %>
    default:
        <%="cumo_#{c_iter}_kernel_dim"%><<<gridDim, blockDim>>>(*a1,*a2,*a3,*indexer);
        break;
    }
}
<% end %>
