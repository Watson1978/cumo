#include "hip/hip_runtime.h"
<% unless is_object %>
__global__ void <%="cumo_#{c_iter}_index_kernel"%>(char *p1, size_t* idx1, seq_data_t beg, seq_data_t step, seq_data_t base, seq_count_t c, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        dtype x = f_seq(beg,step,c+i);
        *(dtype*)(p1+idx1[i]) = m_pow(base,x);
    }
}

__global__ void <%="cumo_#{c_iter}_stride_kernel"%>(char *p1, size_t s1, seq_data_t beg, seq_data_t step, seq_data_t base, seq_count_t c, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        dtype x = f_seq(beg,step,c+i);
        *(dtype*)(p1+(i*s1)) = m_pow(base,x);
    }
}

void <%="cumo_#{c_iter}_index_kernel_launch"%>(char *p1, size_t* idx1, seq_data_t beg, seq_data_t step, seq_data_t base, seq_count_t c, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    <%="cumo_#{c_iter}_index_kernel"%><<<grid_dim, block_dim>>>(p1,idx1,beg,step,base,c,n);
}

void <%="cumo_#{c_iter}_stride_kernel_launch"%>(char *p1, ssize_t s1, seq_data_t beg, seq_data_t step, seq_data_t base, seq_count_t c, uint64_t n)
{
    size_t grid_dim = cumo_get_grid_dim(n);
    size_t block_dim = cumo_get_block_dim(n);
    <%="cumo_#{c_iter}_stride_kernel"%><<<grid_dim, block_dim>>>(p1,s1,beg,step,base,c,n);
}
<% end %>
