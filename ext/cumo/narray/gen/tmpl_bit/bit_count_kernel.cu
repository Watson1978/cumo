#include "hip/hip_runtime.h"
#undef int_t
#define int_t unsigned long long int

__global__ void <%="cumo_#{c_iter}_index_kernel"%>(size_t p1, char* p2, BIT_DIGIT *a1, size_t *idx1, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        BIT_DIGIT x=0;
        CUMO_LOAD_BIT(a1, p1 + idx1[i], x);
        if (m_<%=name%>(x)) {
            atomicAdd((int_t*)p2, (int_t)1);
        }
    }
}

__global__ void <%="cumo_#{c_iter}_stride_kernel"%>(size_t p1, char* p2, BIT_DIGIT *a1, ssize_t s1, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        BIT_DIGIT x=0;
        CUMO_LOAD_BIT(a1, p1 + i * s1, x);
        if (m_<%=name%>(x)) {
            atomicAdd((int_t*)p2, (int_t)1);
        }
    }
}

__global__ void <%="cumo_#{c_iter}_index_stride_kernel"%>(size_t p1, char* p2, BIT_DIGIT *a1, size_t *idx1, ssize_t s2, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        BIT_DIGIT x=0;
        CUMO_LOAD_BIT(a1, p1 + idx1[i], x);
        if (m_<%=name%>(x)) {
            atomicAdd((int_t*)(p2 + i * s2), (int_t)1);
        }
    }
}

__global__ void <%="cumo_#{c_iter}_stride_stride_kernel"%>(size_t p1, char* p2, BIT_DIGIT *a1, ssize_t s1, ssize_t s2, uint64_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        BIT_DIGIT x=0;
        CUMO_LOAD_BIT(a1, p1 + i * s1, x);
        if (m_<%=name%>(x)) {
            atomicAdd((int_t*)(p2 + i * s2), (int_t)1);
        }
    }
}

void <%="cumo_#{c_iter}_index_kernel_launch"%>(size_t p1, char *p2, BIT_DIGIT *a1, size_t *idx1, uint64_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="cumo_#{c_iter}_index_kernel"%><<<gridDim, blockDim>>>(p1,p2,a1,idx1,n);
}

void <%="cumo_#{c_iter}_stride_kernel_launch"%>(size_t p1, char *p2, BIT_DIGIT *a1, ssize_t s1, uint64_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="cumo_#{c_iter}_stride_kernel"%><<<gridDim, blockDim>>>(p1,p2,a1,s1,n);
}

void <%="cumo_#{c_iter}_index_stride_kernel_launch"%>(size_t p1, char *p2, BIT_DIGIT *a1, size_t *idx1, ssize_t s2, uint64_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="cumo_#{c_iter}_index_stride_kernel"%><<<gridDim, blockDim>>>(p1,p2,a1,idx1,s2,n);
}

void <%="cumo_#{c_iter}_stride_stride_kernel_launch"%>(size_t p1, char *p2, BIT_DIGIT *a1, ssize_t s1, ssize_t s2, uint64_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    <%="cumo_#{c_iter}_stride_stride_kernel"%><<<gridDim, blockDim>>>(p1,p2,a1,s1,s2,n);
}

#undef int_t
