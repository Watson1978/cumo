#include "hip/hip_runtime.h"
#include <cstdint>
#include "cumo/template_kernel.h"

#if defined(__cplusplus)
extern "C" {
#if 0
} /* satisfy cc-mode */
#endif
#endif

__global__ void na_parse_array_index_kernel(size_t* idx, ssize_t* nidxp, ssize_t size, size_t n)
{
    for (uint64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        if (nidxp[i] < 0) {
            nidxp[i] += size;
        }
        idx[i] = nidxp[i];
    }
}

void na_parse_array_index_kernel_launch(size_t* idx, ssize_t* nidxp, ssize_t size, size_t n)
{
    size_t gridDim = get_gridDim(n);
    size_t blockDim = get_blockDim(n);
    na_parse_array_index_kernel<<<gridDim, blockDim>>>(idx,nidxp,size,n);
}

#if defined(__cplusplus)
#if 0
{ /* satisfy cc-mode */
#endif
}  /* extern "C" { */
#endif
